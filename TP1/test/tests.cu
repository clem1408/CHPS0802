#include <gtest/gtest.h>
#include <hip/hip_runtime_api.h>
#include "../src/prac1b_impl.h"

TEST(CudaTest, VectorAddition) {
  float *h_v, *h_v2, *h_v3, *d_v, *d_v2, *d_v3;
  int   nblocks, nthreads, nsize, n;

  // Set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // Allocations of the three vectors

  h_v = (float *)malloc(nsize*sizeof(float));
  h_v2 = (float *)malloc(nsize*sizeof(float));
  h_v3 = (float *)malloc(nsize*sizeof(float));

  checkCudaErrors(hipMalloc((void **)&d_v, nsize*sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_v2, nsize*sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_v3, nsize*sizeof(float)));

  // Mallocs gestion

  if (h_v == NULL || h_v2 == NULL || h_v3 == NULL) {
    printf("Erreur d'allocation mémoire\n");
  }

  // Init of the two vectors

  fill_vector_randomly(h_v, nsize);
  fill_vector_randomly(h_v2, nsize);

  // Addition of the two vectors on the cpu

  printf("============================\n");

  add_floats_vectors(h_v, h_v2, h_v3, nsize);

  printf("============================\n");

  // Meccopy of the two vectors

  checkCudaErrors( hipMemcpy(d_v,h_v,nsize*sizeof(float),
                 hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(d_v2,h_v2,nsize*sizeof(float),
                 hipMemcpyHostToDevice) );

  // Execute kernel

  add_vectors_kernel<<<nblocks,nthreads>>>(d_v, d_v2, d_v3);
  getLastCudaError("add_vectors_kernel execution failed\n");

  // Copy back results and print them out

  checkCudaErrors( hipMemcpy(h_v3,d_v3,nsize*sizeof(float),
               hipMemcpyDeviceToHost) );

  for (n=0; n<nsize; n++) printf("[DEVICE] Résultat : %f\n",h_v3[n]);

  printf("============================\n");

  for (n=0; n<nsize; n++) EXPECT_EQ(h_v3[n], h_v[n] + h_v2[n]);

  printf("============================\n");

  // Free memory

  checkCudaErrors(hipFree(d_v));
  checkCudaErrors(hipFree(d_v2));
  checkCudaErrors(hipFree(d_v3));
  free(h_v);
  free(h_v2);
  free(h_v3);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}