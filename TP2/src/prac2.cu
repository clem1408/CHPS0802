#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>



////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N, a = 1, b = 2, c = 3;
__constant__ float T, r, sigma, rho, alpha, dt, con1, con2;



////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////

__global__ void pathcalc(float *d_z, float *d_v)
{
  float s1, s2, y1, y2, payoff;
  int   ind;

  // Move array pointers to correct position

  // Version 1
  ind = threadIdx.x + 2*N*blockIdx.x*blockDim.x;

  // Version 2
  //ind = 2*N*threadIdx.x + 2*N*blockIdx.x*blockDim.x;


  // Path calculation

  s1 = 1.0f;
  s2 = 1.0f;

  for (int n=0; n<N; n++)
  {
    y1   = d_z[ind];

    // Version 1
    ind += blockDim.x;      // Shift pointer to next element
    // Version 2
    // ind += 1;

    y2   = rho*y1 + alpha*d_z[ind];
    // Version 1
    ind += blockDim.x;      // Shift pointer to next element
    // Version 2
    // ind += 1;

    s1 = s1*(con1 + con2*y1);
    s2 = s2*(con1 + con2*y2);
  }

  // Put payoff value into device array

  payoff = 0.0f;
  if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f )
    payoff = exp(-r*T);

  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff;
}

__global__ void generate_random(hiprandState *d_states, float *x, unsigned long seed)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

    hiprand_init(seed, idx, 0, &d_states[idx]);

	hiprandState localState = d_states[idx];

    float sum = 0.0f;

    for (int i = 0; i < 200; i++) {
        float z = hiprand_normal(&localState);  // Generate a z in [0,1]
        sum += a * z * z + b * z + c;
    }

    x[idx] = sum / 200;
    d_states[idx] = localState;
}



////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv)
{
  int     NPATH=9600000, h_N=100, nblocks, nthreads, nsize;
  float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2, h_result;
  float  *h_v, *h_x, *d_v, *d_z, *d_x;
  double  sum1, sum2;

  // Initialise card

  findCudaDevice(argc, argv);

  // Initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);

  checkCudaErrors( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*2*h_N*NPATH) );

  // Define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.05f;
  h_sigma = 0.1f;
  h_rho   = 0.5f;
  h_alpha = sqrt(1.0f-h_rho*h_rho);
  h_dt    = 1.0f/h_N;
  h_con1  = 1.0f + h_r*h_dt;
  h_con2  = sqrt(h_dt)*h_sigma;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(rho),  &h_rho,  sizeof(h_rho)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(alpha),&h_alpha,sizeof(h_alpha)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2)) );

  // Random number generation

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );

  hipEventRecord(start);
  checkCudaErrors( hiprandGenerateNormal(gen, d_z, 2*h_N*NPATH, 0.0f, 1.0f) );
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, 2.0*h_N*NPATH/(0.001*milli));

  // Execute kernel and time it

  hipEventRecord(start);
  pathcalc<<<NPATH/128, 128>>>(d_z, d_v);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  getLastCudaError("pathcalc execution failed\n");
  printf("Monte Carlo kernel execution time (ms): %f \n",milli);

  // Copy back results

  checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );

  // Compute average

  sum1 = 0.0;
  sum2 = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_v[i];
    sum2 += h_v[i]*h_v[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );

  printf("\n========================================================= \n\n");

  // Beginning of computation of the average value of az² + bz + c

  // Set number of blocks, and threads per block

  nblocks  = 256;
  nthreads = 256;
  nsize    = nblocks*nthreads ;
  hiprandState *d_states;

  h_x = (float *)malloc(sizeof(float)*nsize);
  checkCudaErrors(hipMalloc((void **)&d_x, sizeof(float)*nsize));
  hipMalloc(&d_states, nsize * sizeof(hiprandState));

  // Error gestion

  if (h_x == NULL)
  {
    printf("Erreur d'allocation mémoire\n");
    return -1;
  }

  int h_a = 1, h_b = 2, h_c = 3;
  hipMemcpyToSymbol(HIP_SYMBOL(a), &h_a, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(b), &h_b, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(c), &h_c, sizeof(int));

  // Execute the kernel

  generate_random<<<nblocks,nthreads>>>(d_states, d_x, time(NULL));

  // Copy back results and print them out

  checkCudaErrors( hipMemcpy(h_x,d_x,nsize*sizeof(float),
                 hipMemcpyDeviceToHost) );

  float temporary_sum = 0.0f;

  for (int n=0; n<nsize; n++) temporary_sum += h_x[n];

  h_result = temporary_sum/nsize;

  printf("Final result: %f\n\n", h_result);

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  free(h_v);
  free(h_x);
  checkCudaErrors( hipFree(d_v) );
  checkCudaErrors( hipFree(d_z) );
  checkCudaErrors( hipFree(d_x) );

  printf("\n========================================================= \n\n");

  // Calculate bandwidth
  size_t bytes_transferred = (2 * h_N * NPATH * sizeof(float)) + (NPATH * sizeof(float));
  float time_seconds = milli / 1000.0f;
  float bandwidth_GBs = bytes_transferred / (time_seconds * 1e9);

  printf("Effective bandwidth: %f GB/s\n", bandwidth_GBs);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
}