#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float T, r, sigma, rho, alpha, dt, con1, con2;


////////////////////////////////////////////////////////////////////////
// kernel routines -- see sections 3.5, 3.6 in cuRAND documentation
////////////////////////////////////////////////////////////////////////

__global__ void RNG_init(hiprandState *state)
{
  // RNG initialisation with id-based skipahead
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(1234, id, 0, &state[id]);
}


__global__ void pathcalc(hiprandState *device_state, float *d_v,
                         int mpath, int NPATH)
{
  float s1, s2, y1, y2, payoff;

  int id = threadIdx.x + blockIdx.x*blockDim.x;
  hiprandState_t state = device_state[id];

  for(int m=0; m<mpath; m++) {
    s1 = 1.0f;
    s2 = 1.0f;

    for (int n=0; n<N; n++) {
      y1 = hiprand_normal(&state);
      y2 = rho*y1 + alpha*hiprand_normal(&state);

      s1 = s1*(con1 + con2*y1);
      s2 = s2*(con1 + con2*y2);
    }

    // put payoff value into device array

    payoff = 0.0f;
    if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f ) payoff = exp(-r*T);

    int payoff_id = id + m*gridDim.x*blockDim.x;
    if (payoff_id < NPATH) d_v[payoff_id] = payoff;
  }
}



////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv) {
    int     NPATH = 9600000, h_N = 100;
    float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2;
    float  *h_v, *d_v;
    double  sum1, sum2;
    hiprandState *state;

    // Initialise GPU
    findCudaDevice(argc, argv);

    // Initialise CUDA timing
    float milli;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocation mémoire
    h_v = (float *)malloc(sizeof(float) * NPATH);
    hipMalloc((void **)&d_v, sizeof(float) * NPATH);
    hipMalloc((void **)&state, sizeof(hiprandState) * NPATH);

    printf("size of hiprandState is %d bytes\n", sizeof(hiprandState));

    // Définition des constantes
    h_T     = 1.0f;
    h_r     = 0.05f;
    h_sigma = 0.1f;
    h_rho   = 0.5f;
    h_alpha = sqrt(1.0f - h_rho * h_rho);
    h_dt    = 1.0f / h_N;
    h_con1  = 1.0f + h_r * h_dt;
    h_con2  = sqrt(h_dt) * h_sigma;

    hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N));
    hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T));
    hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r));
    hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma));
    hipMemcpyToSymbol(HIP_SYMBOL(rho),  &h_rho,  sizeof(h_rho));
    hipMemcpyToSymbol(HIP_SYMBOL(alpha),&h_alpha,sizeof(h_alpha));
    hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt));
    hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1));
    hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2));

    // Calcul de l'occupation théorique
    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    int maxActiveBlocks, blockSize = 128;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, pathcalc, blockSize, 0);
    printf("maxActiveBlocks/SM = %d \n", maxActiveBlocks);
    printf("number of SMs      = %d \n", props.multiProcessorCount);
    int blocks = maxActiveBlocks * props.multiProcessorCount;

    // Exécution des kernels
    hipEventRecord(start);
    RNG_init<<<blocks, 128>>>(state);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milli, start, stop);

    printf("RNG_init kernel execution time (ms): %f \n", milli);

    int paths_per_thread = (NPATH - 1) / (128 * blocks) + 1;
    hipEventRecord(start);
    pathcalc<<<blocks, 128>>>(state, d_v, paths_per_thread, NPATH);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milli, start, stop);

    printf("pathcalc kernel execution time (ms): %f \n", milli);

    // Copie des résultats vers l'hôte
    hipMemcpy(h_v, d_v, sizeof(float) * NPATH, hipMemcpyDeviceToHost);

    // Calcul de la moyenne et de l'écart type
    sum1 = 0.0;
    sum2 = 0.0;
    for (int i = 0; i < NPATH; i++) {
        sum1 += h_v[i];
        sum2 += h_v[i] * h_v[i];
    }

    printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
           sum1 / NPATH, sqrt((sum2 / NPATH - (sum1 / NPATH) * (sum1 / NPATH)) / NPATH));

    // Libération de la mémoire
    free(h_v);
    hipFree(d_v);
    hipFree(state);

    // CUDA exit -- needed to flush printf write buffer
    hipDeviceReset();
}